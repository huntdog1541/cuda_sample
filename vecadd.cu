#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256

// kernel code for adding two vector elements
__global__ void vecAdd(float* a, float* b, float* c)
{
	int i = threadIdx.x;
	if (i < N)
		c[i] = a[i] + b[i];
}

int main(void)
{
	int   i;
	float a[N], b[N], c[N];
	float *devPtrA, *devPtrB, *devPtrC;

	// initialize arrays
	for (i=0; i < N; i++) {
		a[i] = -i;
		b[i] = i*i;
	}

	// allocate CUDA memory for arrays
	int memsize = N*sizeof(float);
	hipMalloc((void**)&devPtrA, memsize);
	hipMalloc((void**)&devPtrB, memsize);
	hipMalloc((void**)&devPtrC, memsize);

	// copy host data to CUDA memory
	hipMemcpy(devPtrA, a, memsize, hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, b, memsize, hipMemcpyHostToDevice);

	// call add function on CUDA GPU
	vecAdd<<<1, N>>>(devPtrA, devPtrB, devPtrC);

	// copy results back
	hipMemcpy(c, devPtrC, memsize, hipMemcpyDeviceToHost);

	// print results
	for (i=0; i < N; i++)
		printf("C[%d]=%f\n", i, c[i]);

	hipFree(devPtrA);
	hipFree(devPtrB);
	hipFree(devPtrC);
	
	return 0;
}
